/**
Copyright (C) <2023>  <Dezeming>

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "cuda_common.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <string>
#include <iostream>

namespace CudaTools {

void printCudaDevice() {
	COUT_BLUE("Cuda Support Devices") << std::endl;

	hipDeviceProp_t prop;
	int count;
	bool accuracy = Get_CUDA_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		accuracy = Get_CUDA_ERROR(hipGetDeviceProperties(&prop, i));
		if (accuracy) {
			std::cout << "GPU sequence " + std::to_string(i) << std::endl;
			std::cout << "  GPU name " << prop.name << std::endl;
			std::cout << "  Compute capability " << prop.major << "." << prop.minor << std::endl;
			std::cout << "  Clock rate " << prop.clockRate << std::endl;
			std::cout << "  Total global memory " << prop.totalGlobalMem / 1024 / 1024 / 1024 << "GB" << std::endl;
			std::cout << "  Total constant memory " << prop.totalConstMem / 1024 << "KB" << std::endl;
			std::cout << "  Multiprocessor count " << prop.multiProcessorCount << std::endl;
		}

	}
}

std::string getCudaErrorStr(const char* error, const char* file, int line) {
	return (std::string(error) + " in " + std::string(file) + " at line " + std::to_string(line));
	//exit(EXIT_FAILURE);
}

bool getCudaError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		COUT_RED("Error: ") << getCudaErrorStr(hipGetErrorString(err), file, line) << std::endl;
		return false;
	}
	else {
		return true;
	}
}

bool getCudaError(hipError_t err) {
	if (err != hipSuccess) {
		COUT_RED("Error: ") << hipGetErrorString(err) << std::endl;
		return false;
	}
	else {
		return true;
	}
}




}















